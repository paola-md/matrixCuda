
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ 
void mult(int *d_a,int *d_b, int *d_c, int m)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < m && row < m) 
    {
        for(int i = 0; i < m; i++) 
        {
            sum += d_a[row * m + i] * d_b[i * m + col];
        }
        d_c[row * m + col] = sum;
    }
} 



int main(int argc, char* argv[]) {
	
	int m;
    printf("Ingresa el tamaño de la matrix cuadrada \n");
    scanf("%d", &m);
	 //int blockSize = 256;
	//int numBlocks = (N + blockSize - 1) / blockSize;
	
	// Allocate memory space on the device 
	  int *d_a, *d_b, *d_c;
	  // Allocate Unified Memory – accessible from CPU or GPU
	  hipMallocManaged(&d_a, sizeof(int)*m*m);
	  hipMallocManaged(&d_b, sizeof(int)*m*m); 
	  hipMallocManaged(&d_c, sizeof(int)*m*m); 
		
	
	int i, j;
    //initialize matrix A
    for (i = 0; i < m; ++i) {
        for (j = 0; j < m; ++j) {
            d_a[i * m + j] = 1; 
        }
    }

    //initialize matrix B
    for (i = 0; i < m; ++i) {
        for (j = 0; j < m; ++j) {
            d_b[i * m + j] = 2; 
        }
    }


	int blockSize = m*m;
	//mult<<<numBlocks, blockSize>>>(d_a, d_b, d_c, m);
	mult<<<1, blockSize>>>(d_a, d_b, d_c, m);
    
     // Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();	


	printf("\n Resultado \n");
	/* Check for errors (all values should be 3.0f) */ 
	int res = m*2;
	int maxError = 0;
    for (i = 0; i < m; ++i) {
        for (j = 0; j < m; ++j) {
			maxError = fmax(maxError, fabs( d_c[i * m + j]-res));
            printf("%d ", d_c[i * m + j] ); 
        }
		printf("\n");
    }
	
	  printf("Max error: %d \n", maxError);
	
	
	// free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

	return 0;
}