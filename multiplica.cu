
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

/* Kernel function to add the elements of two arrays */
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int i;
  for (i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20; /* 1M elements */
  int i;
  float *x;
  float *y;
  float maxError;
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));  
  
  /* initialize x and y arrays on the host */
  for (i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  /* Run kernel on 1M elements on the GPU */
  add<<<numBlocks, blockSize>>>(N, x, y);
  
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  
  /* Check for errors (all values should be 3.0f) */
  maxError = 0.0f;
  for (i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));

  printf("Max error: %f \n", maxError);

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}
