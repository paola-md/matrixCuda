#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ 
void mult(int *d_a,int *d_b, int *d_c, int m)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;

    if( col < m && row < m) 
    {
        for(int i = 0; i < m; i++) 
        {
            sum += d_a[row * m + i] * d_b[i * m + col];
        }
        d_c[row * m + col] = sum;
    }
} 



int main(int argc, char* argv[]) {
	
	int m;
    printf("Ingresa el tamaño de la matrix cuadrada \n");
    scanf("%d", &m);
	 //int blockSize = 256;
	//int numBlocks = (N + blockSize - 1) / blockSize;
	
	// Allocate memory space on the device 
	  int *d_a, *d_b, *d_c;
	  // Allocate Unified Memory – accessible from CPU or GPU
	  hipMallocManaged(&d_a, sizeof(int)*m*m);
	  hipMallocManaged(&d_b, sizeof(int)*m*m); 
	  hipMallocManaged(&d_c, sizeof(int)*m*m); 
		
	
	int i, j;
    //initialize matrix A
    for (i = 0; i < m; ++i) {
        for (j = 0; j < m; ++j) {
            d_a[i * m + j] = 1; 
        }
    }

    //initialize matrix B
    for (i = 0; i < m; ++i) {
        for (j = 0; j < m; ++j) {
            d_b[i * m + j] = 2; 
        }
    }
	
	// some events to count the execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	
    // start to count execution time of GPU version
    hipEventRecord(start, 0);
	
	int N = m;
	dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);
        if (N*N > 512){
            threadsPerBlock.x = 512;
            threadsPerBlock.y = 512;
            blocksPerGrid.x = ceil(double(N)/double(threadsPerBlock.x));
            blocksPerGrid.y = ceil(double(N)/double(threadsPerBlock.y));
        }
		

    mult<<<blocksPerGrid,threadsPerBlock>>>(d_a, d_b, d_c, m);

    
     // Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();	

    // time counting terminate
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // compute time elapse on GPU computing
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n\n", m, m, m, m, gpu_elapsed_time_ms);

	printf("\n Resultado \n");
	/* Check for errors (all values should be 3.0f) */ 
	int res = m*2;
	int maxError = 0;
    for (i = 0; i < m; ++i) {
        for (j = 0; j < m; ++j) {
			maxError = fmax(maxError, fabs( d_c[i * m + j]-res));
            printf("%d ", d_c[i * m + j] ); 
        }
		printf("\n");
    }
	
	  printf("Max error: %d \n", maxError);
	
	
	// free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

	return 0;
}